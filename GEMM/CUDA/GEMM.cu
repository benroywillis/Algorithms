#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include "TimingLib.h"

#if PRECISION == 0 // float
#define TYPE float
#else // double
#define TYPE double
#endif

#ifndef SIZE
#define SIZE 		64
#endif

#ifndef THREADS_PER_BLOCK
#define THREADS_PER_BLOCK 256
#endif

#ifndef CHECK
#define CHECK 0
#endif

__global__
void GEMM(TYPE *in0, TYPE *in1, TYPE *out)
{
	int stride = blockDim.x * gridDim.x;
	int index  = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = index; i < SIZE; i += stride )
    {
        for (int j = 0; j < SIZE; j++)
        {
            for (int k = 0; k < SIZE; k++)
            {
                out[i*SIZE+j] += in0[i*SIZE+k] * in1[k*SIZE+j];
            }
        }
    }
}

int main()
{
	TYPE* in0, *in1, *out, *out_check;
	hipMallocManaged(&in0, SIZE*SIZE*sizeof(TYPE));
	hipMallocManaged(&in1, SIZE*SIZE*sizeof(TYPE));
	hipMallocManaged(&out, SIZE*SIZE*sizeof(TYPE));
	hipMallocManaged(&out_check, SIZE*SIZE*sizeof(TYPE));

    for (int i = 0; i < SIZE; i++)
    {
        for (int j = 0; j < SIZE; j++)
        {
            in0[i*SIZE+j] = rand();
            in1[i*SIZE+j] = rand();
            out[i*SIZE+j] = 0;
            out_check[i*SIZE+j] = 0;
        }
    }

	__TIMINGLIB_benchmark([&]{ GEMM<<< (SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(in0, in1, out); hipDeviceSynchronize(); });

	// keeps the optimizer from ruining the experiment
	volatile bool yes = out[0];
	/*for( unsigned i = 0; i < SIZE; i++ )
	{
		for( unsigned j = 0; j < SIZE; j++ )
		{
			volatile bool yes = out[i*SIZE+j];
		}
	}*/
#if CHECK == 1
	print("Running check between CUDA answer and naive C answer...\n");
	for( unsigned i = 0; i < SIZE; i++ )
	{
		for( unsigned j = 0; j < SIZE; j++ )
		{
			for( unsigned k = 0; k < SIZE ; k++ )
			{
				out_check[i*SIZE+j] += in0[i*SIZE+k] * in1[k*SIZE+j];
			}
		}
	}
	double error = 0.0;
	double sum   = 0.0;
	for( unsigned i = 0; i < SIZE; i++ )
	{
		for( unsigned j = 0; j < SIZE; j++ )
		{
			error += abs(out[i*SIZE+j]  - out_check[i*SIZE+j]);
			sum   += abs(out[i*SIZE+j]) + abs(out_check[i*SIZE+j]);
		}
	}
	printf("Difference: %g\n", (error / sum) * 100);
#endif

	hipFree(in0);
	hipFree(in1);
	hipFree(out);
	hipFree(out_check);
    return 0;
}
