
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// taken from https://developer.nvidia.com/blog/even-easier-introduction-cuda/

#define THREADS_PER_BLOCK 256

// function that adds two floats, the __global__ macro tells the CUDA compiler this can be run on a GPU
__global__
void add( int n, float* x, float* y )
{
	// CUDA kernels have 1 parameters that form a polyhedron
	// 0: a grid of blocks (x, y, z)
	// 1. a block of threads (x, y, z)
	// gridDim: number of blocks in a Dim
	// blockDim: number of threads in a block
	// blockIdx: the index of the current block
	// threadIdx: the index of the current thread

	// the size of a polyhedron
	int stride = blockDim.x*gridDim.x;
	// the index into the current polyhedron
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	for( int i = index; i < n; i += stride )
	{
		y[i] = x[i] + y[i];
	}
}

int main()
{
	int N = 1 << 20;
	float* x,* y;
	//x = new float[N];
	//y = new float[N];
	// cuda equivalents
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	for( int i = 0; i < N; i++ )
	{
		x[i] = 1.0f;
		y[i] = 2.0f;
	}
	//add( N, x , y );
	// cuda kernel launch indicated by the <<<>>> syntax
	// numbers mean: <<<num thread blocks, num threads in a thread block>>> and are called execution configuration
	add<<< (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>( N, x , y );
	// cuda synchronization code to wait for kernel completion
	hipDeviceSynchronize();

	float maxError = 0.0f;
	for( int i = 0; i < N; i++ )
	{
		maxError = fmax( maxError, fabs(y[i] - 3.0f ) );
	}
	std::cout << "Max error: " << maxError << std::endl;

	//delete [] x;
	//delete [] y;
	// cuda equivalents
	hipFree(x);
	hipFree(y);

	return 0;
}
